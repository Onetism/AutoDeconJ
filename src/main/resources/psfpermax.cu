
#include <hip/hip_runtime.h>
// extern "C"
// __global__ void psfpermax(float *d_Src,float *max_value,
//                         int sizeH1,int sizeH2,int sizeH3,int sizeH4,int sizeH5)
// {
//     const long x = blockIdx.x*blockDim.x+threadIdx.x;

//     long H1 = sizeH1;
//     long H2 = sizeH2;
//     long H3 = sizeH3;
//     long H4 = sizeH4;
//     long H5 = sizeH5;

//     if(x < H1*H2*H3*H4*H5)
//     {
//         d_Src[x] = d_Src[x]/max_value;            
//     }
// }